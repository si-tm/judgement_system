#include "hip/hip_runtime.h"
/*
 * CUDADetailedPolymerSwapInteraction.cu
 *
 *  Created on: 17/mar/2022
 *      Author: lorenzo
 */

#include "CUDADetailedPolymerSwapInteraction.h"

#include "Particles/CustomParticle.h"
#include "CUDA/Lists/CUDASimpleVerletList.h"
#include "CUDA/Lists/CUDANoList.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/transform.h>

#define CUDA_MAX_SWAP_NEIGHS 20

/* System constants */
__constant__ int MD_N[1];
__constant__ int MD_n[1];
__constant__ int MD_interaction_matrix_size[1];
__constant__ float MD_sqr_rep_rcut[1];
__constant__ float MD_sqr_rfene[1];
__constant__ float MD_Kfene[1];
__constant__ float MD_WCA_sigma[1];
__constant__ float MD_sqr_rcut[1];
__constant__ float MD_alpha[1];
__constant__ float MD_beta[1];
__constant__ float MD_gamma[1];

__constant__ float MD_sqr_3b_rcut[1];
__constant__ float MD_3b_sigma[1];
__constant__ float MD_3b_prefactor[1];
__constant__ float MD_3b_rcut[1];
__constant__ float MD_3b_epsilon[1];
__constant__ float MD_3b_A_part[1];
__constant__ float MD_3b_B_part[1];

__constant__ bool MD_enable_semiflexibility[1];
__constant__ float MD_semiflexibility_k[1];

texture<float, 1, hipReadModeElementType> tex_eps;

#include "CUDA/cuda_utils/CUDA_lr_common.cuh"

struct __align__(16) CUDA_FS_bond {
	c_number4 force;
	c_number epsilon;
	int q;
};

struct __align__(16) CUDA_FS_bond_list {
	int n_bonds;
	CUDA_FS_bond bonds[CUDA_MAX_SWAP_NEIGHS];

	__device__
	CUDA_FS_bond_list() :
					n_bonds(0) {
	}
	__device__
	void add_bond(c_number4 &force, c_number epsilon, int q) {
		n_bonds++;
		if(n_bonds > CUDA_MAX_SWAP_NEIGHS) {
			printf("TOO MANY SWAP NEIGHBOURS, TRAGEDY\nHere is the list of neighbours:\n");
			for(int i = 0; i < n_bonds; i++) {
				printf("%d ", bonds[i].q);
			}
			printf("\n");
			// this will invalidate the status of the simulation without crashing it
			n_bonds--;
		}
		bonds[n_bonds - 1].force = force;
		bonds[n_bonds - 1].q = q;
		bonds[n_bonds - 1].epsilon = epsilon;
	}
};

__device__ void _WCA(c_number4 &ppos, c_number4 &qpos, c_number4 &F, CUDABox *box) {
	c_number4 r = box->minimum_image(ppos, qpos);
	c_number sqr_r = CUDA_DOT(r, r);

	c_number energy = 0.f;
	// this is the module of the force over r, so we don't have to divide the distance vector by its module
	c_number force_mod = 0.f;

	if(sqr_r < MD_sqr_rep_rcut[0]) {
		c_number part = 1.f;
		c_number ir2_scaled = SQR(MD_WCA_sigma[0]) / sqr_r;
		for(int i = 0; i < MD_n[0] / 2; i++) {
			part *= ir2_scaled;
		}
		energy += 4.f * part * (part - 1.f) + 1.f - MD_alpha[0];
		force_mod += 4.f * MD_n[0] * part * (2.f * part - 1.f) / sqr_r;
	}
	/*else {
		energy += 0.5f * MD_alpha[0] * (cosf(MD_gamma[0] * sqr_r + MD_beta[0]) - 1.f);
		force_mod += MD_alpha[0] * MD_gamma[0] * sinf(MD_gamma[0] * sqr_r + MD_beta[0]);
	}

	if(sqr_r > MD_sqr_rcut[0]) {
		energy = force_mod = (c_number) 0.f;
		}*/

	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy;
}

__device__ void _sticky(c_number4 &ppos, c_number4 &qpos, int eps_idx, int q_idx, c_number4 &F, CUDA_FS_bond_list &bond_list, CUDABox *box) {
	c_number4 r = box->minimum_image(ppos, qpos);
	c_number sqr_r = CUDA_DOT(r, r);

	c_number energy = 0.f;
	// this c_number is the module of the force over r, so we don't have to divide the distance vector by its module
	c_number force_mod = 0.f;

	if(sqr_r < MD_sqr_3b_rcut[0]) {
		c_number r_mod = sqrtf(sqr_r);
		c_number delta_r = r_mod - MD_3b_rcut[0];
		c_number epsilon = tex1Dfetch(tex_eps, eps_idx);
		// given the finite precision of floating point numbers, this might be equal to or ever-so-slightly larger than 0
		if(delta_r < 0.f && epsilon != 0.f) {
			c_number exp_part = expf(MD_3b_sigma[0] / delta_r);
			c_number tmp_energy = epsilon * MD_3b_A_part[0] * exp_part * (MD_3b_B_part[0] / SQR(sqr_r) - 1.f);
			
			energy += tmp_energy;
			
			force_mod = (epsilon * MD_3b_A_part[0] * exp_part * (4.f * MD_3b_B_part[0] / (SQR(sqr_r) * r_mod)) + MD_3b_sigma[0] * tmp_energy / SQR(delta_r)) / r_mod;

			c_number4 tmp_force = r * force_mod;
			tmp_force.w = (r_mod < MD_3b_sigma[0]) ? epsilon : -tmp_energy;
			
			bond_list.add_bond(tmp_force, epsilon, q_idx);
		}
	}
	
	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy;
}

__device__ void _FENE(c_number4 &ppos, c_number4 &qpos, c_number4 &F, CUDABox *box) {
	c_number sqr_rfene = MD_sqr_rfene[0];
	c_number Kfene = MD_Kfene[0];

	c_number4 r = box->minimum_image(ppos, qpos);
	c_number sqr_r = CUDA_DOT(r, r);

	if(sqr_r > sqr_rfene) {
		printf("WARNING: the distance between particles %d and %d (%lf) exceeds the FENE R0 (%lf)\n", get_particle_index(ppos), get_particle_index(qpos), sqrtf(sqr_r), sqrtf(sqr_rfene));
	}

	c_number energy = -Kfene * sqr_rfene * logf(1.f - sqr_r / sqr_rfene);
	// this c_number is the module of the force over r, so we don't have to divide the distance vector by its module
	c_number force_mod = -2.f * Kfene * sqr_rfene / (sqr_rfene - sqr_r);

	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy;
}

__device__ void _sticky_three_body(CUDA_FS_bond_list &bond_list, c_number4 &F, c_number4 *forces) {
	for(int bi = 0; bi < bond_list.n_bonds; bi++) {
		CUDA_FS_bond &b1 = bond_list.bonds[bi];
		c_number curr_energy = b1.force.w / b1.epsilon;
		
		for(int bj = bi + 1; bj < bond_list.n_bonds; bj++) {
			CUDA_FS_bond &b2 = bond_list.bonds[bj];
			c_number other_energy = b2.force.w / b2.epsilon;

			number smallest_epsilon = min(b1.epsilon, b2.epsilon);
			number prefactor = MD_3b_prefactor[0] * smallest_epsilon;

			// the factor 2 takes into account the fact that the pair energy is always counted twice
			F.w += 2.f * prefactor * curr_energy * other_energy;

			if(curr_energy < 1.f) {
				c_number factor = -prefactor * other_energy;
				c_number4 force = factor * b1.force;
				force.w = 0.f;

				F -= force;
				LR_atomicAddXYZ(forces + b1.q, force);
			}

			if(other_energy < 1.f) {
				c_number factor = -prefactor * curr_energy;
				c_number4 force = factor * b2.force;
				force.w = 0.f;

				F -= force;
				LR_atomicAddXYZ(forces + b2.q, force);
			}
		}
	}
}

__device__ void _flexibility_three_body(c_number4 &ppos, c_number4 &n1_pos, c_number4 &n2_pos, int n1_idx, int n2_idx, c_number4 &F, c_number4 *poss, c_number4 *three_body_forces, CUDABox *box) {
	c_number4 dist_pn1 = box->minimum_image(ppos, n1_pos);
	c_number4 dist_pn2 = box->minimum_image(n2_pos, ppos);

	c_number sqr_dist_pn1 = CUDA_DOT(dist_pn1, dist_pn1);
	c_number sqr_dist_pn2 = CUDA_DOT(dist_pn2, dist_pn2);
	c_number i_pn1_pn2 = 1.f / sqrtf(sqr_dist_pn1 * sqr_dist_pn2);
	c_number cost = CUDA_DOT(dist_pn1, dist_pn2) * i_pn1_pn2;

	c_number cost_n1 = cost / sqr_dist_pn1;
	c_number cost_n2 = cost / sqr_dist_pn2;
	c_number force_mod_n1 = i_pn1_pn2 + cost_n1;
	c_number force_mod_n2 = i_pn1_pn2 + cost_n2;

	F += dist_pn1 * (force_mod_n1 * MD_semiflexibility_k[0]) - dist_pn2 * (force_mod_n2 * MD_semiflexibility_k[0]);
	F.w += MD_semiflexibility_k[0] * (1.f - cost);

	c_number4 n1_force = dist_pn2 * (i_pn1_pn2 * MD_semiflexibility_k[0]) - dist_pn1 * (cost_n1 * MD_semiflexibility_k[0]);
	c_number4 n2_force = dist_pn2 * (cost_n2 * MD_semiflexibility_k[0]) - dist_pn1 * (i_pn1_pn2 * MD_semiflexibility_k[0]);
	LR_atomicAddXYZ(three_body_forces + n1_idx, n1_force);
	LR_atomicAddXYZ(three_body_forces + n2_idx, n2_force);
}

__device__ int get_monomer_type(const c_number4 &r_i) {
	int my_btype = __float_as_int(r_i.w) >> 22;
	return my_btype > 0;
}

__global__ void ps_FENE_flexibility_forces(c_number4 *poss, c_number4 *forces, c_number4 *three_body_forces, int *bonded_neighs, CUDABox *box) {
	if(IND >= MD_N[0]) return;

	c_number4 F = forces[IND];
	c_number4 ppos = poss[IND];

	// the first value of each column is the number of bonded neighbours
	int n_bonded_neighs = bonded_neighs[IND];

	for(int i = 1; i <= n_bonded_neighs; i++) {
		int i_idx = bonded_neighs[MD_N[0] * i + IND];
		c_number4 i_pos = poss[i_idx];

		_FENE(ppos, i_pos, F, box);

		if(MD_enable_semiflexibility[0]) {
			for(int j = i + 1; j <= n_bonded_neighs; j++) {
				int j_idx = bonded_neighs[MD_N[0] * j + IND];
				c_number4 j_pos = poss[j_idx];
				_flexibility_three_body(ppos, i_pos, j_pos, i_idx, j_idx, F, poss, three_body_forces, box);
			}
		}
	}

	forces[IND] = F;
}

__device__ bool _sticky_interaction(int p_btype, int q_btype) {
	return p_btype != DetailedPolymerSwapInteraction::MONOMER && q_btype != DetailedPolymerSwapInteraction::MONOMER;
}

// forces + second step without lists
__global__ void ps_forces(c_number4 *poss, c_number4 *forces, c_number4 *three_body_forces, CUDABox *box) {
	if(IND >= MD_N[0]) return;

	c_number4 F = forces[IND];
	c_number4 ppos = poss[IND];
	int p_btype = get_particle_btype(ppos);

	CUDA_FS_bond_list bonds;

	for(int j = 0; j < MD_N[0]; j++) {
		if(j != IND) {
			c_number4 qpos = poss[j];
			int q_btype = get_particle_btype(qpos);

			_WCA(ppos, qpos, F, box);
			
			if(_sticky_interaction(p_btype, q_btype)) {
				int eps_idx = p_btype * MD_interaction_matrix_size[0] + q_btype;
				_sticky(ppos, qpos, eps_idx, j, F, bonds, box);
			}
		}
	}

	_sticky_three_body(bonds, F, three_body_forces);

	forces[IND] = F;
}

// forces + second step with verlet lists
__global__ void ps_forces(c_number4 *poss, c_number4 *forces, c_number4 *three_body_forces, int *matrix_neighs, int *c_number_neighs, CUDABox *box) {
	if(IND >= MD_N[0]) return;

	c_number4 F = forces[IND];
	c_number4 ppos = poss[IND];

	int num_neighs = c_number_neighs[IND];
	int p_btype = get_particle_btype(ppos);

	CUDA_FS_bond_list bonds;

	for(int j = 0; j < num_neighs; j++) {
		int q_index = matrix_neighs[j * MD_N[0] + IND];

		c_number4 qpos = poss[q_index];
		int q_btype = get_particle_btype(qpos);

		_WCA(ppos, qpos, F, box);
		
		if(_sticky_interaction(p_btype, q_btype)) {
			int eps_idx = p_btype + MD_interaction_matrix_size[0] * q_btype;
			_sticky(ppos, qpos, eps_idx, q_index, F, bonds, box);
		}
	}

	_sticky_three_body(bonds, F, three_body_forces);

	forces[IND] = F;
}

CUDADetailedPolymerSwapInteraction::CUDADetailedPolymerSwapInteraction() :
				DetailedPolymerSwapInteraction() {
	_d_three_body_forces = nullptr;
	_d_bonded_neighs = nullptr;
}

CUDADetailedPolymerSwapInteraction::~CUDADetailedPolymerSwapInteraction() {
	if(_d_bonded_neighs != nullptr) {
		CUDA_SAFE_CALL(hipFree(_d_bonded_neighs));
	}

	if(_d_three_body_forces != nullptr) {
		CUDA_SAFE_CALL(hipFree(_d_three_body_forces));
	}

	if(_d_3b_epsilon != nullptr) {
		CUDA_SAFE_CALL(hipFree(_d_3b_epsilon));
	}
}

void CUDADetailedPolymerSwapInteraction::get_settings(input_file &inp) {
	DetailedPolymerSwapInteraction::get_settings(inp);
}

void CUDADetailedPolymerSwapInteraction::cuda_init(c_number box_side, int N) {
	CUDABaseInteraction::cuda_init(box_side, N);
	DetailedPolymerSwapInteraction::init();

	std::vector<BaseParticle *> particles(_N);
	DetailedPolymerSwapInteraction::allocate_particles(particles);
	int tmp_N_strands;
	DetailedPolymerSwapInteraction::read_topology(&tmp_N_strands, particles);

	CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_forces, N * sizeof(c_number4)));

	int max_n_neighs = 5;
	int n_elems = (max_n_neighs + 1) * _N;
	CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc<int>(&_d_bonded_neighs, n_elems * sizeof(int)));
	std::vector<int> h_bonded_neighs(n_elems);

	for(int i = 0; i < _N; i++) {
		CustomParticle *p = static_cast<CustomParticle *>(particles[i]);
		// start from 1, since the first element will contain the number of bonds
		int nb = 1;
		for(auto q : p->bonded_neighs) {
			if(nb > max_n_neighs) {
				throw oxDNAException("CUDADetailedPolymerSwapInteraction: particle %d has more than %d bonded neighbours", p->index, max_n_neighs);
			}
			h_bonded_neighs[_N * nb + i] = q->index;
			nb++;
		}
		h_bonded_neighs[i] = nb - 1;
	}

	CUDA_SAFE_CALL(hipMemcpy(_d_bonded_neighs, h_bonded_neighs.data(), n_elems * sizeof(int), hipMemcpyHostToDevice));
	for(auto particle: particles) {
		delete particle;
	}

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_n), &_PS_n, sizeof(int)));
	int interaction_matrix_size = _N_attractive_types + 1;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_interaction_matrix_size), &interaction_matrix_size, sizeof(int)));
	COPY_NUMBER_TO_FLOAT(MD_sqr_rep_rcut, _PS_sqr_rep_rcut)
	COPY_NUMBER_TO_FLOAT(MD_sqr_rfene, _sqr_rfene);
	COPY_NUMBER_TO_FLOAT(MD_Kfene, _Kfene);
	COPY_NUMBER_TO_FLOAT(MD_WCA_sigma, _WCA_sigma);
	COPY_NUMBER_TO_FLOAT(MD_sqr_rcut, _sqr_rcut);
	COPY_NUMBER_TO_FLOAT(MD_alpha, _PS_alpha);
	COPY_NUMBER_TO_FLOAT(MD_beta, _PS_beta);
	COPY_NUMBER_TO_FLOAT(MD_gamma, _PS_gamma);
	COPY_NUMBER_TO_FLOAT(MD_sqr_3b_rcut, _sqr_3b_rcut);
	COPY_NUMBER_TO_FLOAT(MD_3b_sigma, _3b_sigma);
	COPY_NUMBER_TO_FLOAT(MD_3b_prefactor, _3b_prefactor);
	COPY_NUMBER_TO_FLOAT(MD_3b_rcut, _3b_rcut);
	COPY_NUMBER_TO_FLOAT(MD_3b_A_part, _3b_A_part);
	COPY_NUMBER_TO_FLOAT(MD_3b_B_part, _3b_B_part);

	CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_3b_epsilon, _3b_epsilon.size() * sizeof(float)));
	std::vector<float> h_3b_epsilon(_3b_epsilon.begin(), _3b_epsilon.end());
	CUDA_SAFE_CALL(hipMemcpy(_d_3b_epsilon, h_3b_epsilon.data(), _3b_epsilon.size() * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipBindTexture(NULL, tex_eps, _d_3b_epsilon, _3b_epsilon.size() * sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_enable_semiflexibility), &_enable_semiflexibility, sizeof(bool)));
	COPY_NUMBER_TO_FLOAT(MD_semiflexibility_k, _semiflexibility_k);
}

void CUDADetailedPolymerSwapInteraction::compute_forces(CUDABaseList *lists, c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_forces, c_number4 *d_torques, LR_bonds *d_bonds, CUDABox *d_box) {
	thrust::device_ptr<c_number4> t_forces = thrust::device_pointer_cast(d_forces);
	thrust::device_ptr<c_number4> t_three_body_forces = thrust::device_pointer_cast(_d_three_body_forces);
	thrust::fill_n(t_three_body_forces, _N, make_c_number4(0, 0, 0, 0));

	ps_FENE_flexibility_forces
		<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
		(d_poss, d_forces, _d_three_body_forces, _d_bonded_neighs, d_box);
	CUT_CHECK_ERROR("ps_FENE_flexibility_forces DetailedPolymerSwap error");

	CUDASimpleVerletList *_v_lists = dynamic_cast<CUDASimpleVerletList *>(lists);
	if(_v_lists != NULL) {
		if(_v_lists->use_edge()) {
			throw oxDNAException("use_edge unsupported by DetailedPolymerSwapInteraction");
		}

		ps_forces
			<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
			(d_poss, d_forces, _d_three_body_forces, _v_lists->d_matrix_neighs, _v_lists->d_number_neighs, d_box);
		CUT_CHECK_ERROR("forces_second_step DetailedPolymerSwap simple_lists error");
	}

	CUDANoList *_no_lists = dynamic_cast<CUDANoList *>(lists);
	if(_no_lists != NULL) {
		ps_forces
			<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
			(d_poss, d_forces, _d_three_body_forces, d_box);
		CUT_CHECK_ERROR("forces_second_step DetailedPolymerSwap no_lists error");
	}

	// add the three body contributions to the two-body forces
	thrust::transform(t_forces, t_forces + _N, t_three_body_forces, t_forces, thrust::plus<c_number4>());

	/*number energy = GpuUtils::sum_c_number4_to_double_on_GPU(d_forces, _N);
	auto energy_string = Utils::sformat("%lf ", energy / _N / 2.);
	*CONFIG_INFO->backend_info += energy_string;*/
}
