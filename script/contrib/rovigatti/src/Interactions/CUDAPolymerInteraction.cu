#include "hip/hip_runtime.h"
/*
 * CUDAPolymerInteraction.cu
 *
 *  Created on: 08/apr/2019
 *      Author: lorenzo
 */

#include "CUDAPolymerInteraction.h"

#include "CUDA/Lists/CUDASimpleVerletList.h"
#include "CUDA/Lists/CUDANoList.h"

/* BEGIN CUDA */

/* System constants */
__constant__ int MD_N[1];
__constant__ int MD_n_forces[1];

__constant__ float MD_sqr_rfene[1];
__constant__ float MD_sqr_rcut[1];
__constant__ float MD_sqr_rep_rcut[1];
__constant__ float MD_Polymer_lambda[1];

#include "CUDA/cuda_utils/CUDA_lr_common.cuh"

__device__ void _nonbonded(c_number4 &r, int int_type, c_number4 &F) {
	c_number sqr_r = CUDA_DOT(r, r);

	c_number energy = 0.f;
	// this c_number is the module of the force over r, so we don't have to divide the distance
	// vector for its module
	c_number force_mod = 0.f;

	if(sqr_r < MD_sqr_rep_rcut[0]) {
		c_number part = powf(1.f / sqr_r, 3.f);
		energy += 4.f * part * (part - 1.f) + 1.f;
		force_mod += 24.f * part * (2.f * part - 1.f) / sqr_r;
	}

	if(int_type == 2) {
		if(sqr_r < MD_sqr_rep_rcut[0]) energy -= MD_Polymer_lambda[0];
		else {
			c_number part = powf(1.f / sqr_r, 3.f);
			energy += 24.f * part * (part - 1.f);
			force_mod += 24.f * MD_Polymer_lambda[0] * part * (2 * part - 1.f) / sqr_r;
		}
	}

	if(sqr_r > MD_sqr_rcut[0]) energy = force_mod = (c_number) 0.f;

	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy;
}

__device__ void _fene(c_number4 &r, c_number4 &F) {
	c_number sqr_r = CUDA_DOT(r, r);

	c_number energy = -15.f * MD_sqr_rfene[0] * logf(1.f - sqr_r / MD_sqr_rfene[0]);

	// this c_number is the module of the force over r, so we don't have to divide the distance
	// vector by its module
	c_number force_mod = -30.f * MD_sqr_rfene[0] / (MD_sqr_rfene[0] - sqr_r);
	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy;
}

__device__ void _particle_particle_bonded_interaction(c_number4 &ppos, c_number4 &qpos, c_number4 &F) {
	c_number4 r = qpos - ppos;
	// bonded interactions are purely repulsive, so we set the int_type to 0 for all pairs
	_nonbonded(r, 0, F);
	_fene(r, F);
}

__device__ void _particle_particle_interaction(c_number4 &ppos, c_number4 &qpos, c_number4 &F, CUDABox *box) {
	int ptype = get_particle_type(ppos);
	int qtype = get_particle_type(qpos);
	int int_type = ptype + qtype;

	c_number4 r = box->minimum_image(ppos, qpos);
	_nonbonded(r, int_type, F);
}

// forces + second step without lists

__global__ void polymer_forces(c_number4 *poss, c_number4 *forces, LR_bonds *bonds, CUDABox *box) {
	if(IND >= MD_N[0]) return;

	c_number4 F = forces[IND];
	LR_bonds bs = bonds[IND];
	c_number4 ppos = poss[IND];

	if(bs.n3 != P_INVALID) {
		c_number4 qpos = poss[bs.n3];
		_particle_particle_bonded_interaction(ppos, qpos, F);
	}

	if(bs.n5 != P_INVALID) {
		c_number4 qpos = poss[bs.n5];
		_particle_particle_bonded_interaction(ppos, qpos, F);
	}

	for(int j = 0; j < MD_N[0]; j++) {
		if(j != IND && bs.n3 != j && bs.n5 != j) {
			c_number4 qpos = poss[j];
			_particle_particle_interaction(ppos, qpos, F, box);
		}
	}

	forces[IND] = F;
}

__global__ void polymer_forces_edge_nonbonded(c_number4 *poss, c_number4 *forces, edge_bond *edge_list, int n_edges, CUDABox *box) {
	if(IND >= n_edges) return;

	c_number4 dF = make_c_number4(0, 0, 0, 0);

	edge_bond b = edge_list[IND];

	// get info for particle 1
	c_number4 ppos = poss[b.from];

	// get info for particle 2
	c_number4 qpos = poss[b.to];

	_particle_particle_interaction(ppos, qpos, dF, box);

	dF.w *= (c_number) 0.5f;

	int from_index = MD_N[0] * (IND % MD_n_forces[0]) + b.from;
	//int from_index = MD_N[0]*(b.n_from % MD_n_forces[0]) + b.from;
	if((dF.x * dF.x + dF.y * dF.y + dF.z * dF.z + dF.w * dF.w) > (c_number) 0.f) LR_atomicAddXYZ(&(forces[from_index]), dF);

	// Allen Eq. 6 pag 3:
	c_number4 dr = box->minimum_image(ppos, qpos); // returns qpos-ppos
	c_number4 crx = _cross(dr, dF);

	dF.x = -dF.x;
	dF.y = -dF.y;
	dF.z = -dF.z;

	int to_index = MD_N[0] * (IND % MD_n_forces[0]) + b.to;
	if((dF.x * dF.x + dF.y * dF.y + dF.z * dF.z + dF.w * dF.w) > (c_number) 0.f) LR_atomicAddXYZ(&(forces[to_index]), dF);
}

// bonded interactions for edge-based approach

__global__ void polymer_forces_edge_bonded(c_number4 *poss, c_number4 *forces, LR_bonds *bonds) {
	if(IND >= MD_N[0]) return;

	c_number4 F0;

	F0.x = forces[IND].x;
	F0.y = forces[IND].y;
	F0.z = forces[IND].z;
	F0.w = forces[IND].w;

	c_number4 dF = make_c_number4(0, 0, 0, 0);
	c_number4 ppos = poss[IND];
	LR_bonds bs = bonds[IND];

	if(bs.n3 != P_INVALID) {
		c_number4 qpos = poss[bs.n3];
		_particle_particle_bonded_interaction(ppos, qpos, dF);
	}
	if(bs.n5 != P_INVALID) {
		c_number4 qpos = poss[bs.n5];
		_particle_particle_bonded_interaction(ppos, qpos, dF);
	}

	forces[IND] = (dF + F0);
}

// forces + second step with verlet lists

__global__ void polymer_forces(c_number4 *poss, c_number4 *forces, int *matrix_neighs, int *c_number_neighs, LR_bonds *bonds, CUDABox *box) {
	if(IND >= MD_N[0]) return;

	c_number4 F = forces[IND];
	c_number4 ppos = poss[IND];
	LR_bonds bs = bonds[IND];

	if(bs.n3 != P_INVALID) {
		c_number4 qpos = poss[bs.n3];
		_particle_particle_bonded_interaction(ppos, qpos, F);
	}
	if(bs.n5 != P_INVALID) {
		c_number4 qpos = poss[bs.n5];
		_particle_particle_bonded_interaction(ppos, qpos, F);
	}

	const int num_neighs = c_number_neighs[IND];
	for(int j = 0; j < num_neighs; j++) {
		const int k_index = matrix_neighs[j * MD_N[0] + IND];

		c_number4 qpos = poss[k_index];
		_particle_particle_interaction(ppos, qpos, F, box);
	}

	forces[IND] = F;
}

/* END CUDA */

CUDAPolymerInteraction::CUDAPolymerInteraction() {

}

CUDAPolymerInteraction::~CUDAPolymerInteraction() {

}

void CUDAPolymerInteraction::get_settings(input_file &inp) {
	PolymerInteraction::get_settings(inp);

	int sort_every;
	if(getInputInt(&inp, "CUDA_sort_every", &sort_every, 0) == KEY_FOUND) {
		if(sort_every > 0) throw oxDNAException("Polymer interaction is not compatible with particle sorting, aborting");
	}
}

void CUDAPolymerInteraction::cuda_init(c_number box_side, int N) {
	CUDABaseInteraction::cuda_init(box_side, N);
	PolymerInteraction::init();

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)));
	COPY_NUMBER_TO_FLOAT(MD_sqr_rfene, this->_sqr_rfene);
	COPY_NUMBER_TO_FLOAT(MD_sqr_rcut, this->_sqr_rcut);
	COPY_NUMBER_TO_FLOAT(MD_sqr_rep_rcut, this->_Polymer_sqr_rep_rcut);
	COPY_NUMBER_TO_FLOAT(MD_Polymer_lambda, this->_Polymer_lambda);

	if(this->_use_edge) CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_n_forces), &this->_n_forces, sizeof(int)));
}

void CUDAPolymerInteraction::compute_forces(CUDABaseList *lists, c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_forces, c_number4 *d_torques, LR_bonds *d_bonds, CUDABox *d_box) {
	CUDASimpleVerletList *_v_lists = dynamic_cast<CUDASimpleVerletList *>(lists);
	if(_v_lists != NULL) {
		if(_v_lists->use_edge()) {
			polymer_forces_edge_nonbonded
				<<<(_v_lists->N_edges - 1)/(this->_launch_cfg.threads_per_block) + 1, this->_launch_cfg.threads_per_block>>>
				(d_poss, this->_d_edge_forces, _v_lists->d_edge_list, _v_lists->N_edges, d_box);

			this->_sum_edge_forces(d_forces);

			// potential for removal here
			hipDeviceSynchronize();
			CUT_CHECK_ERROR("forces_second_step error -- after non-bonded");

			polymer_forces_edge_bonded
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_forces, d_bonds);
		}
		else {
			polymer_forces
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_forces, _v_lists->d_matrix_neighs, _v_lists->d_number_neighs, d_bonds, d_box);
			CUT_CHECK_ERROR("forces_second_step simple_lists error");
		}
	}
	else {
		CUDANoList *_no_lists = dynamic_cast<CUDANoList *>(lists);

		if(_no_lists != NULL) {
			polymer_forces
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss,  d_forces, d_bonds, d_box);
			CUT_CHECK_ERROR("forces_second_step no_lists error");
		}
	}
}
