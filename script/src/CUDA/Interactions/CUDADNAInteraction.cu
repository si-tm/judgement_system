#include "hip/hip_runtime.h"
/*
 * CUDADNAInteraction.cu
 *
 *  Created on: 22/feb/2013
 *      Author: lorenzo
 */

#include "CUDADNAInteraction.h"

#include "CUDA_DNA.cuh"
#include "../Lists/CUDASimpleVerletList.h"
#include "../Lists/CUDANoList.h"
#include "../../Interactions/DNA2Interaction.h"

CUDADNAInteraction::CUDADNAInteraction() {

}

CUDADNAInteraction::~CUDADNAInteraction() {

}

void CUDADNAInteraction::get_settings(input_file &inp) {
	_use_debye_huckel = false;
	_use_oxDNA2_coaxial_stacking = false;
	_use_oxDNA2_FENE = false;
	std::string inter_type;
	if(getInputString(&inp, "interaction_type", inter_type, 0) == KEY_FOUND) {
		if(inter_type.compare("DNA2") == 0) {
			_use_debye_huckel = true;
			_use_oxDNA2_coaxial_stacking = true;
			_use_oxDNA2_FENE = true;

			// we don't need the F4_... terms as the macros are used in the CUDA_DNA.cuh file; this doesn't apply for the F2_K term
			F2_K[1] = CXST_K_OXDNA2;
			_debye_huckel_half_charged_ends = true;
			this->_grooving = true;
			// end copy from DNA2Interaction

			// copied from DNA2Interaction::get_settings() (CPU), the least bad way of doing things
			getInputNumber(&inp, "salt_concentration", &_salt_concentration, 1);
			getInputBool(&inp, "dh_half_charged_ends", &_debye_huckel_half_charged_ends, 0);

			// lambda-factor (the dh length at T = 300K, I = 1.0)
			_debye_huckel_lambdafactor = 0.3616455f;
			getInputFloat(&inp, "dh_lambda", &_debye_huckel_lambdafactor, 0);

			// the prefactor to the Debye-Huckel term
			_debye_huckel_prefactor = 0.0543f;
			getInputFloat(&inp, "dh_strength", &_debye_huckel_prefactor, 0);
			// End copy from DNA2Interaction
		}
	}

	// this needs to be here so that the default value of this->_grooving can be overwritten
	DNAInteraction::get_settings(inp);
}

void CUDADNAInteraction::cuda_init(c_number box_side, int N) {
	CUDABaseInteraction::cuda_init(box_side, N);
	DNAInteraction::init();

	float f_copy = this->_hb_multiplier;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_hb_multi), &f_copy, sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)));

	c_number tmp[50];
	for(int i = 0; i < 2; i++)
		for(int j = 0; j < 5; j++)
			for(int k = 0; k < 5; k++)
				tmp[i * 25 + j * 5 + k] = this->F1_EPS[i][j][k];

	COPY_ARRAY_TO_CONSTANT(MD_F1_EPS, tmp, 50);

	for(int i = 0; i < 2; i++) {
		for(int j = 0; j < 5; j++) {
			for(int k = 0; k < 5; k++) {
				tmp[i * 25 + j * 5 + k] = this->F1_SHIFT[i][j][k];
			}
		}
	}

	COPY_ARRAY_TO_CONSTANT(MD_F1_SHIFT, tmp, 50);

	COPY_ARRAY_TO_CONSTANT(MD_F1_A, this->F1_A, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RC, this->F1_RC, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_R0, this->F1_R0, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_BLOW, this->F1_BLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_BHIGH, this->F1_BHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RLOW, this->F1_RLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RHIGH, this->F1_RHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RCLOW, this->F1_RCLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RCHIGH, this->F1_RCHIGH, 2);

	COPY_ARRAY_TO_CONSTANT(MD_F2_K, this->F2_K, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RC, this->F2_RC, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_R0, this->F2_R0, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_BLOW, this->F2_BLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_BHIGH, this->F2_BHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RLOW, this->F2_RLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RHIGH, this->F2_RHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RCLOW, this->F2_RCLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RCHIGH, this->F2_RCHIGH, 2);

	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_A, this->F5_PHI_A, 4);
	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_B, this->F5_PHI_B, 4);
	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_XC, this->F5_PHI_XC, 4);
	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_XS, this->F5_PHI_XS, 4);

	if(this->_use_edge) CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_n_forces), &this->_n_forces, sizeof(int)));
	if(_use_debye_huckel) {
		// copied from DNA2Interaction::init() (CPU), the least bad way of doing things
		// We wish to normalise with respect to T=300K, I=1M. 300K=0.1 s.u. so divide this->_T by 0.1
		c_number lambda = _debye_huckel_lambdafactor * sqrt(this->_T / 0.1f) / sqrt(_salt_concentration);
		// RHIGH gives the distance at which the smoothing begins
		_debye_huckel_RHIGH = 3.0 * lambda;
		_minus_kappa = -1.0 / lambda;

		// these are just for convenience for the smoothing parameter computation
		c_number x = _debye_huckel_RHIGH;
		c_number q = _debye_huckel_prefactor;
		c_number l = lambda;

		// compute the some smoothing parameters
		_debye_huckel_B = -(exp(-x / l) * q * q * (x + l) * (x + l)) / (-4. * x * x * x * l * l * q);
		_debye_huckel_RC = x * (q * x + 3. * q * l) / (q * (x + l));

		c_number debyecut;
		if(this->_grooving) {
			debyecut = 2.0f * sqrt((POS_MM_BACK1) * (POS_MM_BACK1) + (POS_MM_BACK2) * (POS_MM_BACK2)) + _debye_huckel_RC;
		}
		else {
			debyecut = 2.0f * sqrt(SQR(POS_BACK)) + _debye_huckel_RC;
		}
		// the cutoff radius for the potential should be the larger of rcut and debyecut
		if(debyecut > this->_rcut) {
			this->_rcut = debyecut;
			this->_sqr_rcut = debyecut * debyecut;
		}
		// End copy from DNA2Interaction

		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_RC), &_debye_huckel_RC, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_RHIGH), &_debye_huckel_RHIGH, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_prefactor), &_debye_huckel_prefactor, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_B), &_debye_huckel_B, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_minus_kappa), &_minus_kappa, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_half_charged_ends), &_debye_huckel_half_charged_ends, sizeof(bool)));
	}
}

void CUDADNAInteraction::_on_T_update() {
	cuda_init(_box_side, _N);
}

void CUDADNAInteraction::compute_forces(CUDABaseList*lists, c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_forces, c_number4 *d_torques, LR_bonds *d_bonds, CUDABox*d_box) {
	CUDASimpleVerletList*_v_lists = dynamic_cast<CUDASimpleVerletList*>(lists);
	if(_v_lists != NULL) {
		if(_v_lists->use_edge()) {
			dna_forces_edge_nonbonded
				<<<(_v_lists->N_edges - 1)/(this->_launch_cfg.threads_per_block) + 1, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_orientations, this->_d_edge_forces, this->_d_edge_torques, _v_lists->d_edge_list, _v_lists->N_edges, d_bonds, this->_grooving, _use_debye_huckel, _use_oxDNA2_coaxial_stacking, d_box);

			this->_sum_edge_forces_torques(d_forces, d_torques);

			CUT_CHECK_ERROR("forces_second_step error -- after non-bonded");

			dna_forces_edge_bonded
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_orientations, d_forces, d_torques, d_bonds, this->_grooving, _use_oxDNA2_FENE, this->_use_mbf, this->_mbf_xmax, this->_mbf_finf);
		}
		else {
			dna_forces
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_orientations, d_forces, d_torques, _v_lists->d_matrix_neighs, _v_lists->d_number_neighs, d_bonds, this->_grooving, _use_debye_huckel, _use_oxDNA2_coaxial_stacking, _use_oxDNA2_FENE, this->_use_mbf, this->_mbf_xmax, this->_mbf_finf, d_box);
			CUT_CHECK_ERROR("forces_second_step simple_lists error");
		}
	}

	CUDANoList*_no_lists = dynamic_cast<CUDANoList*>(lists);
	if(_no_lists != NULL) {
			dna_forces
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_orientations,  d_forces, d_torques, d_bonds, this->_grooving, _use_debye_huckel, _use_oxDNA2_coaxial_stacking, _use_oxDNA2_FENE, this->_use_mbf, this->_mbf_xmax, this->_mbf_finf, d_box);
			CUT_CHECK_ERROR("forces_second_step no_lists error");
	}
}

void CUDADNAInteraction::_hb_op_precalc(c_number4 *poss, GPU_quat *orientations, int *op_pairs1, int *op_pairs2, float *hb_energies, int n_threads, bool *region_is_nearhb, CUDA_kernel_cfg _ffs_hb_precalc_kernel_cfg, CUDABox*d_box) {
	hb_op_precalc<<<_ffs_hb_precalc_kernel_cfg.blocks, _ffs_hb_precalc_kernel_cfg.threads_per_block>>>(poss, orientations, op_pairs1, op_pairs2, hb_energies, n_threads, region_is_nearhb, d_box);
	CUT_CHECK_ERROR("hb_op_precalc error");
}

void CUDADNAInteraction::_near_hb_op_precalc(c_number4 *poss, GPU_quat *orientations, int *op_pairs1, int *op_pairs2, bool *nearly_bonded_array, int n_threads, bool *region_is_nearhb, CUDA_kernel_cfg _ffs_hb_precalc_kernel_cfg, CUDABox*d_box) {
	near_hb_op_precalc<<<_ffs_hb_precalc_kernel_cfg.blocks, _ffs_hb_precalc_kernel_cfg.threads_per_block>>>(poss, orientations, op_pairs1, op_pairs2, nearly_bonded_array, n_threads, region_is_nearhb, d_box);
	CUT_CHECK_ERROR("nearhb_op_precalc error");
}

void CUDADNAInteraction::_dist_op_precalc(c_number4 *poss, GPU_quat *orientations, int *op_pairs1, int *op_pairs2, c_number *op_dists, int n_threads, CUDA_kernel_cfg _ffs_dist_precalc_kernel_cfg, CUDABox*d_box) {
	dist_op_precalc<<<_ffs_dist_precalc_kernel_cfg.blocks, _ffs_dist_precalc_kernel_cfg.threads_per_block>>>(poss, orientations, op_pairs1, op_pairs2, op_dists, n_threads, d_box);
	CUT_CHECK_ERROR("dist_op_precalc error");
}
