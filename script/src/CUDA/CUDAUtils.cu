#include "hip/hip_runtime.h"
/*
 * GpuUtils.cpp
 *
 *  Created on: 24/set/2010
 *      Author: lorenzo
 */

#include "CUDAUtils.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>

size_t GpuUtils::_allocated_dev_mem = 0;

__global__ void print_array(int *v, int N) {
	for(int i = 0; i < N; i++)
		printf("%d %d\n", i, v[i]);
}

__global__ void print_array(float *v, int N) {
	for(int i = 0; i < N; i++)
		printf("%d %f\n", i, v[i]);
}

__global__ void print_array(double *v, int N) {
	for(int i = 0; i < N; i++)
		printf("%d %lf\n", i, v[i]);
}

__global__ void print_array(LR_double4 *v, int N) {
	for(int i = 0; i < N; i++)
		printf("%d %lf %lf %lf %lf\n", i, v[i].x, v[i].y, v[i].z, v[i].w);
}

__global__ void print_array(float4 *v, int N) {
	for(int i = 0; i < N; i++)
		printf("%d %lf %lf %lf %lf\n", i, v[i].x, v[i].y, v[i].z, v[i].w);
}

template<typename T>
__global__ void check_thresold(T *v, int N, int t) {
	for(int i = 0; i < N; i++)
		if(v[i] >= t) printf("%d %d\n", i, v[i]);
}

template<typename T>
void GpuUtils::print_device_array(T *v, int N) {
print_array
		<<<1,1>>>
		(v, N);
		CUT_CHECK_ERROR("print_device_array error");
	hipDeviceSynchronize();
}

template<typename T>
void GpuUtils::check_device_thresold(T *v, int N, int t) {
check_thresold<T>
		<<<1,1>>>
		(v, N, t);
		CUT_CHECK_ERROR("check_device_thresold error");
	hipDeviceSynchronize();
}

struct sum_c_number4 {
	__device__
	c_number4 operator()(const c_number4& a, const c_number4& b) const {
		c_number4 res;
		res.x = a.x + b.x;
		res.y = a.y + b.y;
		res.z = a.z + b.z;
		res.w = a.w + b.w;
		return res;
	}
};

struct c_number4_to_double {
	__device__
	double operator()(const c_number4 &a) {
		return (double) a.w;
	}
};

c_number4 GpuUtils::sum_c_number4_on_GPU(c_number4 *dv, int N) {
	thrust::device_ptr<c_number4> t_dv = thrust::device_pointer_cast(dv);
	c_number4 zero = { 0., 0., 0., 0. };
	return thrust::reduce(t_dv, t_dv + N, zero, sum_c_number4());
}

double GpuUtils::sum_c_number4_to_double_on_GPU(c_number4 *dv, int N) {
	thrust::device_ptr<c_number4> t_dv = thrust::device_pointer_cast(dv);
	return thrust::transform_reduce(t_dv, t_dv + N, c_number4_to_double(), 0., thrust::plus<double>());
}
