#include "hip/hip_runtime.h"
/*
 * CUDABussiThermostat.cpp
 *
 *  Created on: Feb 15, 2013
 *      Author: rovigatti
 */

#include "CUDABussiThermostat.h"

#include "../../Utilities/ConfigInfo.h"

#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>

struct sum_K {
	__device__
	c_number4 operator()(const c_number4& a, const c_number4& b) const {
		c_number4 res;
		res.x = res.y = res.z = 0;
		res.w = a.w + b.w;
		return res;
	}
};

struct compute_K {
	__device__
	c_number4 operator()(const c_number4& a) const {
		c_number4 res;
		res.x = res.y = res.z = 0;
		res.w = 0.5f * CUDA_DOT(a, a);
		return res;
	}
};

struct compute_K_without_com {
	c_number4 v_com;

	compute_K_without_com(c_number4 my_v_com) : v_com(my_v_com) {

	}

	__device__
	c_number4 operator()(const c_number4& v) const {
		c_number4 v_rel = v - v_com;
		c_number4 res;
		res.x = res.y = res.z = 0;
		res.w = 0.5f * CUDA_DOT(v_rel, v_rel);
		return res;
	}
};


__global__ void bussi_thermostat(c_number4 *vels, c_number4 *Ls, c_number4 v_com, c_number rescale_factor_t, c_number rescale_factor_r, int N) {
	if(IND >= N) return;

	c_number4 v = vels[IND];
	v.x = (v.x - v_com.x) * rescale_factor_t + v_com.x;
	v.y = (v.y - v_com.y) * rescale_factor_t + v_com.y;
	v.z = (v.z - v_com.z) * rescale_factor_t + v_com.z;
	v.w = (v.x * v.x + v.y * v.y + v.z * v.z) * (c_number) 0.5f;
	vels[IND] = v;

	c_number4 L = Ls[IND];
	L.x *= rescale_factor_r;
	L.y *= rescale_factor_r;
	L.z *= rescale_factor_r;
	L.w = (L.x * L.x + L.y * L.y + L.z * L.z) * (c_number) 0.5f;
	Ls[IND] = L;
}

CUDABussiThermostat::CUDABussiThermostat() :
				CUDABaseThermostat(),
				BussiThermostat() {

}

CUDABussiThermostat::~CUDABussiThermostat() {

}

void CUDABussiThermostat::get_settings(input_file &inp) {
	BussiThermostat::get_settings(inp);
	CUDABaseThermostat::get_cuda_settings(inp);
}

void CUDABussiThermostat::init() {
	BussiThermostat::init();

	_setup_rand(CONFIG_INFO->N());
}

bool CUDABussiThermostat::would_activate(llint curr_step) {
	return (curr_step % _newtonian_steps == 0);
}

void CUDABussiThermostat::apply_cuda(c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_vels, c_number4 *d_Ls, llint curr_step) {
	if(!would_activate(curr_step)) return;

	int N = CONFIG_INFO->N();

	// we first calculate the current kinetic energy
	thrust::device_ptr<c_number4> t_vels = thrust::device_pointer_cast(d_vels);
	thrust::device_ptr<c_number4> t_Ls = thrust::device_pointer_cast(d_Ls);

	c_number4 zero = { 0., 0., 0., 0. };
	c_number4 v_com = thrust::reduce(t_vels, t_vels + N, zero);
	v_com *= 1. /  N;
	c_number4 K_now_t = thrust::transform_reduce(t_vels, t_vels + N, compute_K_without_com(v_com), zero, sum_K());
	c_number4 K_now_r = thrust::transform_reduce(t_Ls, t_Ls + N, compute_K(), zero, sum_K());

	_update_K(_K_t, _current_translational_degrees_of_freedom());
	_update_K(_K_r, _current_rotational_degrees_of_freedom());

	c_number rescale_factor_t = std::sqrt(_K_t / K_now_t.w);
	c_number rescale_factor_r = std::sqrt(_K_r / K_now_r.w);

	bussi_thermostat
		<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
		(d_vels, d_Ls, v_com, rescale_factor_t, rescale_factor_r, N);
}
